#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "filters.cuh"
#include "image.cuh"
#include "kernels.cuh"
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define CHANNELS 3

void gaussianBlurCUDA(Image* src, Image* dst, int radius, float sigma) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Calculate Gaussian kernel on CPU
    int kernelSize = 2 * radius + 1;
    float* kernel = (float*)malloc(kernelSize * kernelSize * sizeof(float));
    float sum = 0.0f;
    
    for(int y = -radius; y <= radius; y++) {
        for(int x = -radius; x <= radius; x++) {
            float value = exp(-(x*x + y*y)/(2*sigma*sigma));
            kernel[(y+radius) * kernelSize + (x+radius)] = value;
            sum += value;
        }
    }
    
    // Normalize kernel
    for(int i = 0; i < kernelSize * kernelSize; i++) {
        kernel[i] /= sum;
    }
    
    // Allocate device memory
    unsigned char *d_src, *d_dst;
    float *d_kernel;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    size_t kernel_size = kernelSize * kernelSize * sizeof(float);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMalloc(&d_kernel, kernel_size);
    
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    gaussianKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius,
        sigma,
        d_kernel
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    free(kernel);
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_kernel);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void embossCUDAoptimize(Image* src, Image* dst, float intensity) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    // Calculate shared memory size
    const int radius = 1;
    const int tile_w = BLOCK_DIM_X + 2 * radius;
    const int tile_h = BLOCK_DIM_Y + 2 * radius;
    const size_t smem_size = tile_w * tile_h * src->channels * sizeof(unsigned char);
    
    // Check shared memory size against device limits
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (smem_size > prop.sharedMemPerBlock) {
        printf("Error: Required shared memory (%lu bytes) exceeds device limit (%lu bytes)\n", 
               smem_size, prop.sharedMemPerBlock);
        return;
    }
    
    // Set up grid and block dimensions
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid(
        (src->width + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
        (src->height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y
    );
    
    // Launch kernel with dynamic shared memory size
    embossKernelOptimized<<<grid, block, smem_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        intensity
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

// void embossCUDAoptimize(Image* src, Image* dst, float intensity) {
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start);
    
//     unsigned char *d_src, *d_dst;
//     size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
//     hipMalloc(&d_src, size);
//     hipMalloc(&d_dst, size);
//     hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
//     int total_elements = src->width * src->height * src->channels;
//     int block_size = 1024;
//     int num_blocks = (total_elements + block_size - 1) / block_size;
    
//     embossKernelOptimized<<<num_blocks, block_size>>>(
//         d_src,
//         d_dst,
//         src->width,
//         src->height,
//         src->channels,
//         intensity
//     );
    
//     hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
//     hipEventRecord(stop);
//     hipEventSynchronize(stop);
    
//     float milliseconds = 0;
//     hipEventElapsedTime(&milliseconds, start, stop);
//     printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
//     hipFree(d_src);
//     hipFree(d_dst);
//     hipEventDestroy(start);
//     hipEventDestroy(stop);
// }

void embossCUDA(Image* src, Image* dst, float intensity) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    embossKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        intensity
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipFree(d_src);
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void erosionCUDAoptimize(Image* src, Image* dst, int radius) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    // Calculate shared memory size based on actual radius
    const int tile_w = BLOCK_DIM_X + 2 * radius;
    const int tile_h = BLOCK_DIM_Y + 2 * radius;
    const size_t smem_size = tile_w * tile_h * src->channels * sizeof(unsigned char);
    
    // Check if shared memory size exceeds device limits
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (smem_size > prop.sharedMemPerBlock) {
        printf("Error: Required shared memory (%lu bytes) exceeds device limit (%lu bytes)\n", 
               smem_size, prop.sharedMemPerBlock);
        return;
    }
    
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid(
        (src->width + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
        (src->height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y
    );
    
    // Launch kernel with dynamic shared memory size
    erosionKernelOptimized<<<grid, block, smem_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

void erosionCUDA(Image* src, Image* dst, int radius) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    erosionKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipFree(d_src);
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void dilationCUDAoptimize(Image* src, Image* dst, int radius) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    // Calculate shared memory size based on actual radius
    const int tile_w = BLOCK_DIM_X + 2 * radius;
    const int tile_h = BLOCK_DIM_Y + 2 * radius;
    const size_t smem_size = tile_w * tile_h * src->channels * sizeof(unsigned char);
    
    // Check if shared memory size exceeds device limits
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (smem_size > prop.sharedMemPerBlock) {
        printf("Error: Required shared memory (%lu bytes) exceeds device limit (%lu bytes)\n", 
               smem_size, prop.sharedMemPerBlock);
        return;
    }
    
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid(
        (src->width + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
        (src->height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y
    );
    
    // Launch kernel with dynamic shared memory size
    dilationKernelOptimized<<<grid, block, smem_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

void dilationCUDA(Image* src, Image* dst, int radius) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start event
    hipEventRecord(start);
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    dilationKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

void waveCUDA(Image* src, Image* dst, float amplitudeX, float amplitudeY, float frequencyX, float frequencyY) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start event
    hipEventRecord(start);
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    waveKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        amplitudeX,
        amplitudeY,
        frequencyX,
        frequencyY
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

void oilPaintingCUDAoptimize(Image* src, Image* dst, int radius, int intensityLevels) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start event
    hipEventRecord(start);
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    // Calculate shared memory size based on actual radius
    const int tile_w = BLOCK_DIM_X + 2 * radius;
    const int tile_h = BLOCK_DIM_Y + 2 * radius;
    const size_t smem_size = tile_w * tile_h * src->channels * sizeof(unsigned char);
    
    // Check if shared memory size exceeds device limits
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (smem_size > prop.sharedMemPerBlock) {
        printf("Error: Required shared memory (%lu bytes) exceeds device limit (%lu bytes)\n", 
               smem_size, prop.sharedMemPerBlock);
        return;
    }
    
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid(
        (src->width + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
        (src->height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y
    );
    
    // Launch kernel with dynamic shared memory size
    oilPaintingKernelOptimized<<<grid, block, smem_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius,
        intensityLevels
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

void oilPaintingCUDA(Image* src, Image* dst, int radius, int intensityLevels) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start event
    hipEventRecord(start);
    unsigned char *d_src, *d_dst;
    size_t size = src->width * src->height * src->channels * sizeof(unsigned char);
    
    hipMalloc(&d_src, size);
    hipMalloc(&d_dst, size);
    hipMemcpy(d_src, src->data, size, hipMemcpyHostToDevice);
    
    int total_elements = src->width * src->height * src->channels;
    int block_size = 1024;
    int num_blocks = (total_elements + block_size - 1) / block_size;
    
    oilPaintingKernel<<<num_blocks, block_size>>>(
        d_src,
        d_dst,
        src->width,
        src->height,
        src->channels,
        radius,
        intensityLevels
    );
    
    hipMemcpy(dst->data, d_dst, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time: %.4f seconds\n", milliseconds/1000.0f);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_src);
    hipFree(d_dst);
}

